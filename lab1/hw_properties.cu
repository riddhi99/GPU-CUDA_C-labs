#include<stdio.h>
#include<hip/hip_runtime.h>

int main()

{

    int devCount;

    hipGetDeviceCount(&devCount);

    printf("CUDA Device Query...\n");

    printf("There are %d CUDA devices.\n", devCount);

 
    for (int i = 0; i < devCount; ++i)
    {

        // Get device properties

        printf("\nCUDA Device #%d\n", i);

        hipDeviceProp_t devProp;

        hipGetDeviceProperties(&devProp, i);

        printf("Device Name: %s\n", devProp.name);
        printf("Total Global Memory: %d\n", devProp.totalGlobalMem);
	printf("Maximum Threads per Block: %d\n", devProp.maxThreadsPerBlock);
	printf("Maximum Threads Dimension in X-axis: %d\n", devProp.maxThreadsDim[0]);
	printf("Maximum Threads Dimension in Y-axis: %d\n", devProp.maxThreadsDim[1]);
	printf("Maximum Threads Dimension in Z-axis: %d\n", devProp.maxThreadsDim[2]);
	printf("Maximum Grid Size in X-axis: %d\n", devProp.maxGridSize[0]);
	printf("Maximum Grid Size in Y-axis: %d\n", devProp.maxGridSize[1]);
	printf("Maximum Grid Size in Z-axis: %d\n", devProp.maxGridSize[2]);
	printf("Warp Size: %d\n", devProp.warpSize);
	printf("Clock Rate: %d\n", devProp.clockRate);
	printf("Shared Memory Per Block: %d\n", devProp.sharedMemPerBlock);
	printf("Registers Per Block: %d\n", devProp.regsPerBlock);
	printf("Maximum pitch in bytes allowed by memory copies: %d\n", devProp.memPitch);
	printf("Total Constant Memory: %d\n", devProp.totalConstMem);
	printf("Major compute capability: %d\n", devProp.major);
	printf("Minor compute capability: %d\n", devProp.minor);
	printf("Alignment required for textures: %d\n", devProp.textureAlignment);
	printf("Device can concurrently copy memory and execute a kernel: %d\n", devProp.deviceOverlap);
	printf("Number of multiprocessors on device: %d\n", devProp.multiProcessorCount);
	printf("Specified whether there is a run time limit on kernels: %d\n", devProp.kernelExecTimeoutEnabled);
	printf("Device is integrated as opposed to discrete: %d\n", devProp.integrated);
	printf("Device can map host memory with hipHostAlloc/hipHostGetDevicePointer: %d\n", devProp.canMapHostMemory);
	printf("Compute mode: %d\n", devProp.computeMode);
	printf("Device can possibly execute multiple kernels concurrently: %d\n", devProp.concurrentKernels);
	printf("Device has ECC support enabled: %d\n", devProp.ECCEnabled);
	printf("PCI bus ID of the device: %d\n", devProp.pciBusID);
	printf("PCI device ID of the device: %d\n", devProp.pciDeviceID);
	printf("1 if device is a Tesla device using TCC driver, 0 otherwise: %d\n", devProp.tccDriver);
    }

    return 0;

}
