#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

__global__ void image_conversion(unsigned char *colorImage, unsigned char *grayImage, long long imageWidth, long long imageHeight) 
{
  	int x = threadIdx.x + blockIdx.x * blockDim.x; 
  	
  	int y = threadIdx.y + blockIdx.y * blockDim.y; 

	int idx = x*blockDim.y + y;

	if(idx<imageWidth*imageHeight)
	{
		int r, g, b;
		r = colorImage[3*idx];
		g = colorImage[3*idx + 1];
		b = colorImage[3*idx + 2];
			
		grayImage[idx] = (unsigned char)((21*r + 71*g + 7*b)/100);
		//grayImage[idx] = (pixel);
		//grayImage[3*idx+1] = grayImage[3*idx+2] = grayImage[3*idx] = pixel;
	}
}

int main( int argc, char* argv[] )
{
	FILE *fptr = fopen("parallel_image_conversion.txt", "w");
	//fprintf(fptr, "imageHeight x imageWidth \t Time(milli) \n", );

	int i;

	for (i = 7; i < 28; i++)
	{	

		unsigned char *colorImage_cpu;
		unsigned char *grayImage_cpu;

		char header[100];
		long long imageWidth, imageHeight, ccv;

		char filename[50];
		snprintf(filename, sizeof(filename), "Lenna_%d.ppm", i); 

		FILE *color = fopen(filename, "rb");

		fscanf(color, "%s\n%lld %lld\n%lld\n", header, &imageWidth, &imageHeight, &ccv);

		size_t bytes = imageWidth*imageHeight*sizeof(unsigned char);

		colorImage_cpu = (unsigned char*)malloc(bytes*3);
		grayImage_cpu = (unsigned char*)malloc(bytes);

		FILE *gray = fopen("gray.ppm", "wb");
		fprintf(gray, "P5\n%d %d\n255\n", imageWidth, imageHeight);
		fread(colorImage_cpu, sizeof(unsigned char), imageWidth*imageHeight*3, color);

		fclose(color);
		
		//for(int j=0; j<imageWidth*imageHeight*3; j++)
		//	printf("%d ", colorImage_cpu[j]);

		unsigned char *colorImage_gpu;
		unsigned char *grayImage_gpu;
		
		hipMalloc(&colorImage_gpu, bytes*3);
		hipMalloc(&grayImage_gpu, bytes);

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipMemcpy(colorImage_gpu, colorImage_cpu, bytes*3, hipMemcpyHostToDevice);
		hipMemcpy(grayImage_gpu, grayImage_cpu, bytes, hipMemcpyHostToDevice);

		dim3 blocksize(32, 32);
		dim3 gridsize((int)ceil((float)imageHeight/32.0),(int)ceil((float)imageWidth/32.0));

		//printf("%d %d\n",(int)ceil((float)imageHeight/32),(int)ceil((float)imageWidth/32));
		
		hipEventRecord(start);
		image_conversion<<<gridsize, blocksize>>>(colorImage_gpu, grayImage_gpu, imageWidth, imageHeight);
		hipEventRecord(stop);

		hipEventSynchronize(stop);

		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);

		hipMemcpy( grayImage_cpu, grayImage_gpu, bytes, hipMemcpyDeviceToHost );
	 
	    	hipFree(colorImage_gpu);
	    	hipFree(grayImage_gpu);
	 
		fprintf(fptr, "%d %ldx%ld %lf\n", i, imageHeight, imageWidth, milliseconds);

		//for(int j=0; j<imageWidth*imageHeight; j++)
		//	printf("%hhu ", grayImage_cpu[j]);

		fwrite(grayImage_cpu, sizeof(unsigned char), imageWidth*imageHeight, gray); 
	 	fclose(gray);
		free(colorImage_cpu);
	    	free(grayImage_cpu);
	}

	fclose(fptr);

	return 0;
}
