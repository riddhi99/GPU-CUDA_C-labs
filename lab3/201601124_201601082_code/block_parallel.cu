#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void image_conversion(unsigned char *colorImage, unsigned char *grayImage, int imageWidth, int imageHeight) 
{
	
  	int x = threadIdx.x + blockIdx.x * blockDim.x; 
  	
  	int y = threadIdx.y + blockIdx.y * blockDim.y; 

	int idx = x*blockDim.y + y;

	if(idx<imageWidth*imageHeight)
	{
	  	int r,g,b;
		r = colorImage[3*idx];
		g = colorImage[3*idx+1];
		b = colorImage[3*idx+2];
	  	
		grayImage[idx] = (unsigned char)((21*r + 71*g + 7*b)/100);
	}
}

int main( int argc, char* argv[] )
{
	FILE *fptr = fopen("block_parallel_image_conversion.txt", "w");
	//fprintf(fptr, "imageHeight x imageWidth \t Time(milli) \n", );

	int i, j;

	for (i = 7; i < 28; i++)
	{	

		for(j=5; j<11; j++)
		{

			unsigned char *colorImage_cpu;
			unsigned char *grayImage_cpu;

			char header[100];
			long long imageWidth, imageHeight, ccv;

			char filename[50];
			snprintf(filename, sizeof(filename), "Lenna_%d.ppm", i); 

			FILE *color = fopen(filename, "rb");

			fscanf(color, "%s\n%lld %lld\n%lld\n", header, &imageWidth, &imageHeight, &ccv);

			size_t bytes = imageWidth*imageHeight*sizeof(unsigned char);

			colorImage_cpu = (unsigned char*)malloc(bytes*3);
			grayImage_cpu = (unsigned char*)malloc(bytes);

			fread(colorImage_cpu, sizeof(unsigned char), imageWidth*imageHeight*3, color);

			fclose(color);

			unsigned char *colorImage_gpu;
			unsigned char *grayImage_gpu;
			
			hipMalloc(&colorImage_gpu, bytes*3);
			hipMalloc(&grayImage_gpu, bytes);

			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);

			hipMemcpy(colorImage_gpu, colorImage_cpu, bytes*3, hipMemcpyHostToDevice);
			hipMemcpy(grayImage_gpu, grayImage_cpu, bytes, hipMemcpyHostToDevice);

			int x, y;
			x = pow(2, floor((float)j/2.0));
			y = pow(2, ceil((float)j/2.0));
			dim3 blocksize(x, y);
			dim3 gridsize((int)ceil((float)imageHeight/(float)x),(int)ceil((float)imageWidth/(float)y));

			//blocksize = 1024;
			//gridsize = (int)ceil((float)(imageWidth*imageHeight)/blocksize);

			//printf("here\n");

			hipEventRecord(start);
			image_conversion<<<gridsize, blocksize>>>(colorImage_gpu, grayImage_gpu, imageWidth, imageHeight);
			hipEventRecord(stop);

			hipEventSynchronize(stop);

			float milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);

			hipMemcpy( grayImage_cpu, grayImage_gpu, bytes, hipMemcpyDeviceToHost );
		 
		    	hipFree(colorImage_gpu);
		    	hipFree(grayImage_gpu);
		 
			fprintf(fptr, "%d %d %dx%d %ldx%ld %lf\n", i, j, x, y, imageWidth, imageHeight, milliseconds);
		     
		 	free(colorImage_cpu);
		    	free(grayImage_cpu);
		}
	}

	fclose(fptr);

	return 0;
}
