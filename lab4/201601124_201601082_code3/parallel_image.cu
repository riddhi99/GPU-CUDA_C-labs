
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#define CHANNELS 3
#define in_COMPONENT_COLOR 255
#define RGB_COMPONENT_COLOR 255
#define HALF_WIDTH 4

typedef struct {
     unsigned char red,green,blue;
} PPMPixel;

typedef struct {
     int x, y;
     PPMPixel *data;
} PPMImage;


__global__ void colCon(unsigned char * outImg, unsigned char *inImage, int width, int height, int pwidth, int sz) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  
  int n=sz*2+1;
//int sa=n*n;
  int r[51], g[51], b[51];
  //int *arr;
  if(x<width && y<height){
    int idxI = (y+sz) * pwidth + (x+sz);
    int filIdx=idxI;
    idxI=idxI-((pwidth)*(sz)+(sz));
    for(int i=0;i<n;i++){
      for(int j=0; j<n; j++){
        int idx=i*n+j;
//         arr[idx]=(int)(256*256*inImage[CHANNELS*idxI]+256*inImage[CHANNELS*idxI+1]+
// inImage[CHANNELS*idxI+2]);
        r[idx]=(int)inImage[CHANNELS*idxI];
        g[idx]=(int)inImage[CHANNELS*idxI+1];
        b[idx]=(int)inImage[CHANNELS*idxI+2];
        idxI++;
      }
      idxI+=pwidth-n-1;
    }

    for(int i=0;i<n*n-1;i++){
      for(int j=0;j<n*n-i-1;j++){
        // if(arr[j]>arr[j+1]){
        //   int temp=arr[j];
        //   arr[j]=arr[j+1];
        //   arr[j+1]=temp;
        if(r[j]>r[j+1]){
          int temp=r[j];
          r[j]=r[j+1];
          r[j+1]=temp;
        } 
        if(g[j]>g[j+1]){
          int temp=g[j];
          g[j]=g[j+1];
          g[j+1]=temp;
        } 
        if(b[j]>b[j+1]){
          int temp=b[j];
          b[j]=b[j+1];
          b[j+1]=temp;
        } 
      }
    }
    // int mid=(n*n)/2;
    // outImg[CHANNELS*filIdx+2]=arr[mid]%256;
    // arr[mid]=arr[mid]>>8;
    // outImg[CHANNELS*filIdx+1]=arr[mid]%256;
    // arr[mid]=arr[mid]>>8;
    // outImg[CHANNELS*filIdx]=arr[mid]%256;
    outImg[CHANNELS*filIdx]=(unsigned char)r[(n*n)/2];
    outImg[CHANNELS*filIdx+1]=(unsigned char)g[(n*n)/2];
    outImg[CHANNELS*filIdx+2]=(unsigned char)b[(n*n)/2];
    
  }
}

static PPMImage *readPPM(const char *filename)
{
  char buff[16];
  PPMImage *img;
  FILE *fp;
  int c, rgb_comp_color;
  //open PPM file for reading
  fp = fopen(filename, "rb");
  if (!fp) {
  fprintf(stderr, "Unable to open file '%s'\n", filename);
  exit(1);
  }
  //read image format
  if (!fgets(buff, sizeof(buff), fp)) {
  perror(filename);
  exit(1);
  }
  //check the image format
  if (buff[0] != 'P' || buff[1] != '6') {
  fprintf(stderr, "Invalid image format (must be 'P6')\n");
  exit(1);
  }
  //alloc memory form image
  img = (PPMImage *)malloc(sizeof(PPMImage));
  if (!img) {
  fprintf(stderr, "Unable to allocate memory\n");
  exit(1);
  }
  //check for comments9
  c = getc(fp);
  while (c == '#') {
  while (getc(fp) != '\n') ;
  c = getc(fp);
  }
  ungetc(c, fp);
  //read image size information
  if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
  fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
  exit(1);
  }
  //read rgb component
  if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
  fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
  exit(1);
  }
  //check rgb component depth
  if (rgb_comp_color!= RGB_COMPONENT_COLOR) {
  fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
  exit(1);
  }
  while (fgetc(fp) != '\n') ;
  //memory allocation for pixel data
  img->data = (PPMPixel*)malloc(img->x * img->y * sizeof(PPMPixel));
  if (!img) {
  fprintf(stderr, "Unable to allocate memory\n");
  exit(1);
  }
  //read pixel data from file
  if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
  fprintf(stderr, "Error loading image '%s'\n", filename);
  exit(1);
  }
  fclose(fp);
  return img;
}

void writePPM(const char *filename, PPMImage *img)
{
FILE *fp;
//open file for output
fp = fopen(filename, "wb");
if (!fp) {
fprintf(stderr, "Unable to open file '%s'\n", filename);
exit(1);
}
//write the header file
//image format
fprintf(fp, "P6\n");
//image size
fprintf(fp, "%d %d\n",img->x,img->y);
// rgb component depth
fprintf(fp, "%d\n",RGB_COMPONENT_COLOR);
// pixel data
fwrite(img->data, 3 * img->x, img->y, fp);
fclose(fp);
}

PPMImage  *padImage(PPMImage *img,int size){
  PPMImage *paddedImg = (PPMImage *)malloc(sizeof(PPMImage));
  int h = (img->y+2*size);
  int w = (img->x+2*size);
  printf("size: %d, h: %d, w: %d\n", size,h,w);
  paddedImg->data = (PPMPixel*)malloc(h * w * sizeof(PPMPixel));
  paddedImg->x = w;
  paddedImg->y = h;
  memset(paddedImg->data, 0, h * w * sizeof(unsigned char));
  printf("Image Initialized\n");
   for(int i=0; i<img->y; i++){
       for(int j=0; j<img->x; j++){
       int idxP=(i+size)*w+(j+size);
       int idx=i*img->x+j;
       //printf("i: %d, j: %d, idx: %d, idxP: %d\n",i,j,idx,idxP);
      paddedImg->data[idxP]= img->data[idx];
      //paddedImg->data[idxP].green = img->data[idx].green;
      //paddedImg->data[idxP].blue = img->data[idx].blue;
      }
   }
   for(int j=0;j<size;j++ ){
     for(int i=0;i<paddedImg->y;i++){
       int idxP=i*w+j;
       paddedImg->data[idxP]=paddedImg->data[i*w+size];
    }
   }

   for(int j=paddedImg->x-size ; j< paddedImg->x;j++){
     for(int i=0;i<paddedImg->y;i++){
       int idxP=i*w+j;
        paddedImg->data[idxP]=paddedImg->data[i*w-size-1];
    }
   }

   for(int i=0;i<size;i++){
     for(int j=0;j<paddedImg->x;j++){
       int idxP=i*w+j;
      paddedImg->data[idxP]=paddedImg->data[(size)*w+j];
    }
   }

   for(int i=(paddedImg->y-size);i<(paddedImg->y);i++){

     for(int j=0;j<paddedImg->x;j++){
       int idxP=i*w+j;
       paddedImg->data[idxP]=paddedImg->data[(paddedImg->y-size-1)*w+j];
     }
   }
   return paddedImg;
}

int main(int argc, char **argv){

    PPMImage *image,*filtered,*padedImg;
    unsigned char *in, *out;
    int bytes, sz, n;
    float elapsed;
    
    sz= atoi(argv[1]);
    printf("Size Initialized\n");
    n = 2*sz+1;

    image = readPPM(argv[2]);
    printf("Before padding: h: %d w: %d\n", image->y, image->x);
    padedImg=padImage(image,sz);
    bytes = padedImg->x * padedImg->y * 3;
    out = (unsigned char*)malloc(bytes * sizeof(unsigned char));
    printf("After padding: h: %d w: %d", padedImg->y, padedImg->x);

    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipMalloc(&in, bytes * sizeof(unsigned char));
    hipMalloc(&out, bytes * sizeof(unsigned char));

    hipMemcpy(in,padedImg->data,bytes,hipMemcpyHostToDevice);

    dim3 gridSize((padedImg->x-1)/16+1,(padedImg->y-1)/16+1,1);
    dim3 blockSize(16,16,1);

    hipEventRecord(start);
    colCon<<<gridSize,blockSize>>>(out,in,image->x,image->y,padedImg->x,sz);
    hipEventRecord(stop);
    
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsed, start, stop);
    printf(",%f\n", elapsed);

    filtered = (PPMImage *)malloc(sizeof(PPMImage));
    filtered->x = padedImg->x;
    filtered->y = padedImg->y;
    filtered->data = (PPMPixel*)malloc(bytes * sizeof(PPMPixel));
    
    hipMemcpy(filtered->data,out,bytes,hipMemcpyDeviceToHost);

    writePPM("medianOutput.ppm",filtered);
    //printf("conversion complete");
    hipFree(in);
    hipFree(out);
    return 0;

  }





